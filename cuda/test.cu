
#include <hip/hip_runtime.h>
#include <stdio.h>

// 打印CUDA版本信息的函数
void printCudaVersion() {
    hipError_t err = hipSuccess;
    int runtimeVersion = 0;
    err = hipRuntimeGetVersion(&runtimeVersion);
    if (err == hipSuccess) {
        printf("CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 1000) / 10);
    } else {
        printf("Error getting runtime version: %s\n", hipGetErrorString(err));
    }

    int driverVersion = 0;
    err = hipDriverGetVersion(&driverVersion);
    if (err == hipSuccess) {
        printf("CUDA Driver Version: %d.%d\n", driverVersion / 1000, (driverVersion % 1000) / 10);
    } else {
        printf("Error getting driver version: %s\n", hipGetErrorString(err));
    }
}


__global__ void myKernel() 
{
    printf("Hello, world from the device!\n"); 
} 

int main() 
{ 
    printCudaVersion();
    myKernel<<<4,4>>>(); 
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipError_t));
        return 1;
        }
        else {
            printf("No CUDA error\n");
            }
     hipDeviceSynchronize();
} 

